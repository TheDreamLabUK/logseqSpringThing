
#include <hip/hip_runtime.h>
// Node data structure matching Rust's NodeData
struct NodeData {
    float position[3];    // 12 bytes
    unsigned char mass;   // 1 byte
    unsigned char flags;  // 1 byte
    unsigned char padding[2]; // 2 bytes padding
};

// Velocity data structure matching Rust's VelocityData
struct VelocityData {
    float x;
    float y;
    float z;
};

extern "C" __global__ void compute_forces(
    NodeData* nodes,
    VelocityData* velocities,
    unsigned long long unused,
    unsigned int num_nodes,
    float spring_strength,
    float spring_length,
    float repulsion,
    float attraction,
    float damping
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_nodes) return;

    // Load node data
    NodeData node_i = nodes[idx];
    float3 pos_i = make_float3(
        node_i.position[0],
        node_i.position[1],
        node_i.position[2]
    );
    float mass_i = (float)node_i.mass;
    float3 force = make_float3(0.0f, 0.0f, 0.0f);

    __shared__ float3 shared_positions[256];
    __shared__ float shared_masses[256];

    // Process nodes in tiles to maximize shared memory usage
    for (int tile = 0; tile < (num_nodes + blockDim.x - 1) / blockDim.x; tile++) {
        int shared_idx = tile * blockDim.x + threadIdx.x;
        
        // Load tile into shared memory
        if (shared_idx < num_nodes) {
            NodeData shared_node = nodes[shared_idx];
            shared_positions[threadIdx.x] = make_float3(
                shared_node.position[0],
                shared_node.position[1],
                shared_node.position[2]
            );
            shared_masses[threadIdx.x] = (float)shared_node.mass;
        }
        __syncthreads();

        // Compute forces between current node and all nodes in tile
        #pragma unroll 8
        for (int j = 0; j < blockDim.x && tile * blockDim.x + j < num_nodes; j++) {
            if (tile * blockDim.x + j == idx) continue;

            // Skip nodes with inactive flag
            if ((nodes[tile * blockDim.x + j].flags & 0x1) == 0) continue;

            float3 pos_j = shared_positions[j];
            float mass_j = shared_masses[j];
            
            // Calculate displacement vector
            float3 diff = make_float3(
                pos_i.x - pos_j.x,
                pos_i.y - pos_j.y,
                pos_i.z - pos_j.z
            );

    // Calculate distance with larger minimum clamp to prevent instability
    float dist = fmaxf(sqrtf(diff.x * diff.x + diff.y * diff.y + diff.z * diff.z), 1.0f);
    
    // Scale repulsion force by sqrt of masses to reduce excessive force
    float mass_factor = sqrtf(mass_i * mass_j);
    float force_mag = repulsion * mass_factor / (dist * dist);

    // Add spring force if nodes are connected (check flags)
    if ((node_i.flags & 0x2) && (nodes[tile * blockDim.x + j].flags & 0x2)) {
        // Use spring_length parameter for natural length
        float spring_force = spring_strength * (dist - spring_length);
        // Add attraction component
        force_mag += spring_force * attraction;
    }

    // Clamp maximum force to prevent instability
    force_mag = fminf(force_mag, 1000.0f);

            // Accumulate force
            force.x += force_mag * diff.x / dist;
            force.y += force_mag * diff.y / dist;
            force.z += force_mag * diff.z / dist;
        }
        __syncthreads();
    }

    // Load current velocity
    float3 vel = make_float3(
        velocities[idx].x,
        velocities[idx].y,
        velocities[idx].z
    );

    // Update velocity with damping
    vel.x = (vel.x + force.x) * damping;
    vel.y = (vel.y + force.y) * damping;
    vel.z = (vel.z + force.z) * damping;

    // Update position
    pos_i.x += vel.x;
    pos_i.y += vel.y;
    pos_i.z += vel.z;

    // Store updated position and velocity
    nodes[idx].position[0] = pos_i.x;
    nodes[idx].position[1] = pos_i.y;
    nodes[idx].position[2] = pos_i.z;
    velocities[idx].x = vel.x;
    velocities[idx].y = vel.y;
    velocities[idx].z = vel.z;

    // Flags and mass remain unchanged
}
